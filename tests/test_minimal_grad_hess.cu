#include "hip/hip_runtime.h"
#include <cstdio>
#include <array>
#include "cudual/kernels.cuh"
using namespace cudadual;
struct F3 { template <class Num> CDUAL_HD Num operator()(const std::array<Num,3>& x) const { return sin(x[0]*x[1]) + exp(x[0]) * (x[2]*x[2]*x[2]); } };
int main(){ using T=double; constexpr int N=3; const int batch=1; T x[N]={1.0,0.5,2.0};
  T *dX=nullptr,*df=nullptr,*dg=nullptr,*dH=nullptr; hipMalloc(&dX,sizeof(T)*batch*N);
  hipMalloc(&df,sizeof(T)*batch); hipMalloc(&dg,sizeof(T)*batch*N); hipMalloc(&dH,sizeof(T)*batch*N*N);
  hipMemcpy(dX,x,sizeof(T)*batch*N,hipMemcpyHostToDevice);
  F3 f; launch_grad_hess<T,N>(f,dX,batch,df,dg,dH); hipDeviceSynchronize();
  T f0=0; hipMemcpy(&f0,df,sizeof(T),hipMemcpyDeviceToHost);
  printf("f = %+.6f\n", f0);
  hipFree(dX); hipFree(df); hipFree(dg); hipFree(dH); return 0; }