#include "hip/hip_runtime.h"
#include <cstdio>
#include <array>
#include <vector>
#include <cmath>
#include "cudual/kernels.cuh"
#include "cudual/cudual.cuh"
using namespace cudadual;
struct LG { template <class Num> CDUAL_HD Num operator()(const std::array<Num,1>& x) const { return lgamma(x[0]); } double host_eval(double v) const { return ::lgamma(v); } };
int main(){ using T=double; constexpr int N=1; const int batch=1; T x0=3.25;
  T *dX=nullptr,*df=nullptr,*dg=nullptr,*dH=nullptr;
  hipMalloc(&dX,sizeof(T)*batch*N); hipMalloc(&df,sizeof(T)*batch);
  hipMalloc(&dg,sizeof(T)*batch*N); hipMalloc(&dH,sizeof(T)*batch*N*N);
  hipMemcpy(dX,&x0,sizeof(T),hipMemcpyHostToDevice);
  LG f; launch_grad_hess<T,N>(f,dX,batch,df,dg,dH); hipDeviceSynchronize();
  T f_ad=0,g_ad=0,H_ad=0; hipMemcpy(&f_ad,df,sizeof(T),hipMemcpyDeviceToHost);
  hipMemcpy(&g_ad,dg,sizeof(T),hipMemcpyDeviceToHost); hipMemcpy(&H_ad,dH,sizeof(T),hipMemcpyDeviceToHost);
  auto feval=[&](T v){ return f.host_eval(v); }; T h=1e-6; T g_fd=(feval(x0+h)-feval(x0-h))/(2*h); T H_fd=(feval(x0+h)-2*feval(x0)+feval(x0-h))/(h*h);
  printf("lgamma: grad err=%.3e hess err=%.3e\n", fabs(g_ad-g_fd), fabs(H_ad-H_fd));
  bool ok = fabs(g_ad-g_fd)<1e-6 && fabs(H_ad-H_fd)<1e-4; printf("RESULT: %s\n", ok? "OK":"FAIL");
  hipFree(dX); hipFree(df); hipFree(dg); hipFree(dH); return ok?0:1; }