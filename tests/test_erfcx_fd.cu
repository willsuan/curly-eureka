#include "hip/hip_runtime.h"
#include <cstdio>
#include <array>
#include <cmath>
#include "cudual/kernels.cuh"
#include "cudual/cudual.cuh"
using namespace cudadual;
struct EX { template <class Num> CDUAL_HD Num operator()(const std::array<Num,1>& x) const { return erfcx(x[0]); } double host_eval(double v) const { return std::exp(v*v)*erfc(v); } };
int main(){ using T=double; constexpr int N=1; const int batch=1; T x0=0.75;
  T *dX=nullptr,*df=nullptr,*dg=nullptr,*dH=nullptr; hipMalloc(&dX,sizeof(T)*batch*N); hipMalloc(&df,sizeof(T)*batch);
  hipMalloc(&dg,sizeof(T)*batch*N); hipMalloc(&dH,sizeof(T)*batch*N*N); hipMemcpy(dX,&x0,sizeof(T),hipMemcpyHostToDevice);
  EX f; launch_grad_hess<T,N>(f,dX,batch,df,dg,dH); hipDeviceSynchronize();
  T f_ad=0,g_ad=0,H_ad=0; hipMemcpy(&f_ad,df,sizeof(T),hipMemcpyDeviceToHost);
  hipMemcpy(&g_ad,dg,sizeof(T),hipMemcpyDeviceToHost); hipMemcpy(&H_ad,dH,sizeof(T),hipMemcpyDeviceToHost);
  auto feval=[&](T v){ return f.host_eval(v); }; T h=1e-6; T g_fd=(feval(x0+h)-feval(x0-h))/(2*h);
  T H_fd=(feval(x0+h)-2*feval(x0)+feval(x0-h))/(h*h);
  printf("erfcx: grad err=%.3e hess err=%.3e\n", fabs(g_ad-g_fd), fabs(H_ad-H_fd));
  bool ok=(fabs(g_ad-g_fd)<1e-7)&&(fabs(H_ad-H_fd)<1e-5); printf("RESULT: %s\n", ok? "OK":"FAIL");
  hipFree(dX); hipFree(df); hipFree(dg); hipFree(dH); return ok?0:1; }